#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2020-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "owl/common/math/random.h"

namespace dvr {

  extern "C" __constant__ LaunchParams optixLaunchParams;

  typedef owl::common::LCG<4> Random;
  
  inline __device__
  vec3f backGroundColor()
  {
    const vec2i pixelID = owl::getLaunchIndex();
    const float t = pixelID.y / (float)optixGetLaunchDimensions().y;
    const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    return c;
  }

  inline __device__ vec4f over(const vec4f &A, const vec4f &B)
  {
    return A + (1.f-A.w)*B;
  }
  

  inline __device__
  bool intersect(const Ray &ray,
                 const box3f &box,
                 float &t0,
                 float &t1)
  {
    vec3f lo = (box.lower - ray.origin) / ray.direction;
    vec3f hi = (box.upper - ray.origin) / ray.direction;
    
    vec3f nr = min(lo,hi);
    vec3f fr = max(lo,hi);

    t0 = max(ray.tmin,reduce_max(nr));
    t1 = min(ray.tmax,reduce_min(fr));

    return t0 < t1;
  }

  inline __device__
  vec4f sampleVolume(const vec3f &pos,
                     vec3f &gradient)
  {
    auto &lp = optixLaunchParams;
    const box3f bounds = lp.volume.domain;
    
    vec3f tc
      = (pos - bounds.lower)
      / bounds.size();
    // if (tc.x < 0.f ||
    //     tc.y < 0.f ||
    //     tc.z < 0.f ||
    //     tc.x > 1.f ||
    //     tc.y > 1.f ||
    //     tc.z > 1.f)
    //   return vec4f(0.f);

    // load raw scalar field value from texture
    float value = tex3D<float>(lp.volume.texture,tc.x,tc.y,tc.z);

    // re-map to [0,1] in specified transfer function domain
    const range1f xfDomain = lp.transferFunc.domain;
    if (value < xfDomain.lower ||
        value > xfDomain.upper) {
      gradient = 0.f;
      return vec4f(1.f,1.f,1.f,0.f);
    }
    float remapped
      = (value - xfDomain.lower) / (xfDomain.upper - xfDomain.lower);
    
    float4 xf = tex2D<float4>(lp.transferFunc.texture,remapped,0.5f);
    xf.w *= lp.transferFunc.opacityScale;


    if (xf.w > 1e-4f) {
      const vec3f delta = lp.render.gradientDelta;
      gradient = vec3f(+tex3D<float>(lp.volume.texture,tc.x+delta.x,tc.y,tc.z)
                       -tex3D<float>(lp.volume.texture,tc.x-delta.x,tc.y,tc.z),
                       +tex3D<float>(lp.volume.texture,tc.x,tc.y+delta.y,tc.z)
                       -tex3D<float>(lp.volume.texture,tc.x,tc.y-delta.y,tc.z),
                       +tex3D<float>(lp.volume.texture,tc.x,tc.y,tc.z+delta.z)
                       -tex3D<float>(lp.volume.texture,tc.x,tc.y,tc.z-delta.z));
      gradient = gradient / (length(gradient + 1e-4f));
    } else gradient = 0.f;

    return xf;
  }

  inline __device__
  float firstSampleT(const range1f &rayInterval,
                     const float dt,
                     const float ils_t0)
  {
    float numSegsf = floor((rayInterval.lower - dt*ils_t0)/dt);
    float t = dt * (ils_t0 + numSegsf);
    if (t < rayInterval.lower) t += dt;
    return t;
  }
               
  inline __device__
  vec4f integrateRay(const Ray &ray,
                     const float ils_t0,
                     float &z)
  {
    auto &lp = optixLaunchParams;

    const box3f bounds = lp.volume.domain;
    float t0, t1;
    vec3f color = 0.f;
    float alpha = 0.f;
    if (!intersect(ray,bounds,t0,t1)) {
      z = 1e10f;
      return vec4f(0.f);
    }

    
    const float dt = lp.render.dt;
    for (float t = firstSampleT({t0,t1},dt,ils_t0); t < t1 && alpha < .99f; t += dt) {
      vec3f gradient;
      vec4f sample = sampleVolume(ray.origin+t*ray.direction,gradient);
      sample.w *= 1.f;
      color += dt * (1.f-alpha) * sample.w * vec3f(sample)
        * (.1f+.9f*fabsf(dot(gradient,ray.direction)));
      alpha += dt * (1.f-alpha)*sample.w;
    }

    z = t0;
    return vec4f(color,alpha);
  }
  
  inline __device__ Ray generateRay(const vec2f screen)
  {
    auto &lp = optixLaunchParams;
    vec3f org = lp.camera.org;
    vec3f dir
      = lp.camera.dir_00
      + screen.u * lp.camera.dir_du
      + screen.v * lp.camera.dir_dv;
    dir = normalize(dir);
    if (fabs(dir.x) < 1e-5f) dir.x = 1e-5f;
    if (fabs(dir.y) < 1e-5f) dir.y = 1e-5f;
    if (fabs(dir.z) < 1e-5f) dir.z = 1e-5f;
    return Ray(org,dir,0.f,1e10f);
  }

#if USER_GEOM_METHOD
  inline __device__
  void intersectBox(const box3f &box,
                    const vec3f &org,
                    const vec3f &dir,
                    float &t0, float &t1)
  {
    vec3f lower = (box.lower - org) / dir;
    vec3f upper = (box.upper - org) / dir;
    vec3f nr = min(lower,upper);
    vec3f fr = max(lower,upper);
    t0 = max(t0,reduce_max(nr));
    t1 = min(t1,reduce_min(fr));
  }
  
  OPTIX_INTERSECT_PROGRAM(SpaceSkipperIsec)()
  {
    const SSGeom &self = owl::getProgramData<SSGeom>();
    SSPRD &prd = owl::getPRD<SSPRD>();
    int primID = optixGetPrimitiveIndex();
    box3f box = self.activeBoxes[primID];
    float t0 = optixGetRayTmin();
    float t1 = optixGetRayTmax();
    const vec3f org = optixGetWorldRayOrigin();
    const vec3f dir = optixGetWorldRayDirection();
    intersectBox(box,org,dir,t0,t1);

    if (t1 <= t0) 
      return;
    
    prd.t0 = t0;
    prd.t1 = t1;
    prd.primID = primID;
    optixReportIntersection(t0,primID);
  }
  OPTIX_CLOSEST_HIT_PROGRAM(SpaceSkipperCH)()
  {
    // compute a semi-sueful normal, mostly for debugging
    const SSGeom &self = owl::getProgramData<SSGeom>();
    SSPRD &prd = owl::getPRD<SSPRD>();
    box3f box = self.activeBoxes[prd.primID];
    const vec3f org = optixGetWorldRayOrigin();
    const vec3f dir = optixGetWorldRayDirection();
    vec3f enterPos = org + prd.t0 * dir;
    enterPos = (enterPos - box.lower) / box.size();
    enterPos = 2.f*enterPos - 1.f;
    prd.Ng = 0;
    prd.Ng[arg_max(abs(enterPos))] = 1;
  }
  OPTIX_BOUNDS_PROGRAM(SpaceSkipperBounds)(const void *geomData,
                                           box3f &primBounds,
                                           const int primID)
  {
    const SSGeom &self = *(const SSGeom *)geomData;
    primBounds = self.activeBoxes[primID];
  }
#else
  OPTIX_CLOSEST_HIT_PROGRAM(SpaceSkipperCH)()
  {
    SSPRD &prd = owl::getPRD<SSPRD>();
    prd.t = optixGetRayTmax();
    prd.primID = optixGetPrimitiveIndex();
    const SSGeom &self = owl::getProgramData<SSGeom>();
    vec3i index = self.indices[prd.primID];
    vec3f A = self.vertices[index.x];
    vec3f B = self.vertices[index.y];
    vec3f C = self.vertices[index.z];
    prd.Ng = normalize(cross(B-A,C-A));
#if SS_SINGLE_RAY
    prd.backFace = (optixGetHitKind() == OPTIX_HIT_KIND_TRIANGLE_BACK_FACE);
#endif
  }
#endif

  inline __device__ vec3f hue_to_rgb(float hue)
  {
    float s = saturate( hue ) * 6.0f;
    float r = saturate( fabsf(s - 3.f) - 1.0f );
    float g = saturate( 2.0f - fabsf(s - 2.0f) );
    float b = saturate( 2.0f - fabsf(s - 4.0f) );
    return vec3f(r, g, b); 
  }
    
  inline __device__ vec3f temperature_to_rgb(float t)
  {
    float K = 4.0f / 6.0f;
    float h = K - K * t;
    float v = .5f + 0.5f * t;
    return v * hue_to_rgb(h);
  }
    
                                    
  inline __device__
  vec3f heatMap(float t)
  {
    return temperature_to_rgb(t);
  }
  
  OPTIX_RAYGEN_PROGRAM(renderFrame)()
  {
    auto &lp = optixLaunchParams;
    const int spp = lp.render.spp; 
    const vec2i threadIdx = owl::getLaunchIndex();
    Ray ray = generateRay(vec2f(threadIdx)+vec2f(.5f));

    vec4f bgColor = vec4f(backGroundColor(),1.f);
    Random random(threadIdx.x,threadIdx.y);

    bool useSpaceSkipping
      =  ((lp.render.spaceSkipMode%3) == 0)
      || ((lp.render.spaceSkipMode%3) == 1 && threadIdx.x >= owl::getLaunchDims().x/2);
    uint64_t clock_begin = clock();

    vec4f accumColor = 0.f;
    float z;
    for (int sampleID=0;sampleID<spp;sampleID++) {
      const float ils_t0 = random();

      vec4f color;
      if (lp.render.showBoxes) {
        color = integrateRay(ray,ils_t0,z);
        color = over(color,bgColor);
        
        SSPRD prd = { -1, -1 };
        owl::traceRay(lp.render.ssGeom,ray,prd,
                      OPTIX_RAY_FLAG_DISABLE_ANYHIT|
#if 1
                      // show only front side:
                      OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES
#else
                      // show only back side:
                      OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES
#endif
                      );
        vec4f box_color
          = prd.primID == -1
          ? vec4f(1,1,1,1)
          : vec4f(randomColor(prd.primID/2)
                  //vec3f(.1,.7f,.1f)
                  *vec3f(.2f+.6f*fabsf(dot(ray.direction,prd.Ng)))
                  ,.6);
        color = over(box_color,color);
      }
      else {
        if (useSpaceSkipping) {
          color = vec4f(0.f);
          float last_t_end = 0.f;
          while (1) {
#if USER_GEOM_METHOD
            SSPRD prd;
            prd.primID = -1;
            ray.tmin = last_t_end;
            ray.tmax = 1e10f;
            owl::traceRay(lp.render.ssGeom,ray,prd,
                          OPTIX_RAY_FLAG_DISABLE_ANYHIT);
            if (prd.primID < 0)
              // no hit at all
              break;
            ray.tmin = prd.t0;
            ray.tmax = prd.t1;
            vec4f thisSeg_color = integrateRay(ray,ils_t0,z);
            color = over(color,thisSeg_color);
            if (color.w >= .98f)
              break;

            last_t_end = prd.t1*(1.f+1e-6f);
#elif SS_SINGLE_RAY
            SSPRD prd = { -1, -1 };
            ray.tmin = last_t_end;
            ray.tmax = 1e10f;
            float thisSeg_t0=last_t_end;
            owl::traceRay(lp.render.ssGeom,ray,prd,
                          OPTIX_RAY_FLAG_DISABLE_ANYHIT);
            if (prd.primID < 0)
              // no hit at all
              break;
            float thisSeg_t1=prd.t;
            if (prd.backFace) {
              // this is a EXIT hit, so we must be inside
            }  else {
              // this is a ENTRY hit, so work _starts_ there
              thisSeg_t0 = prd.t;
              ray.tmin = prd.t*(1.f+1e-6f);
              ray.tmax = 1e10f;
              prd.primID = -1;
              owl::traceRay(lp.render.ssGeom,ray,prd,
                            OPTIX_RAY_FLAG_DISABLE_ANYHIT);
              if (prd.primID < 0)
                // could not find exit face!? error ...
                break;
              thisSeg_t1 = prd.t;
            }
            ray.tmin = thisSeg_t0;
            ray.tmax = thisSeg_t1;
            vec4f thisSeg_color = integrateRay(ray,ils_t0,z);
            color = over(color,thisSeg_color);
            if (color.w >= .98f)
              break;

            last_t_end = thisSeg_t1*(1.f+1e-6f);
#else
            SSPRD prd = { -1, -1 };
            ray.tmin = last_t_end;
            ray.tmax = 1e10f;
            owl::traceRay(lp.render.ssGeom,ray,prd,
                          OPTIX_RAY_FLAG_DISABLE_ANYHIT|
                          // show only back side:
                          OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES
                          );
            if (prd.primID < 0) break;
            const float thisSeg_t1 = prd.t;

            prd = { -1, -1 };
            Ray backRay = Ray(ray.origin+thisSeg_t1*ray.direction,
                              -ray.direction,0.f,thisSeg_t1-last_t_end);
            owl::traceRay(lp.render.ssGeom,backRay,prd,
                          OPTIX_RAY_FLAG_DISABLE_ANYHIT|
                          // show only back side:
                          OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES
                          );
            float thisSeg_t0
              = (prd.primID < 0)
              ? last_t_end
              : (thisSeg_t1-prd.t);
            ray.tmin = thisSeg_t0;
            ray.tmax = thisSeg_t1;
            // careful - this overwrites z if there's more than one segment
            vec4f thisSeg_color = integrateRay(ray,ils_t0,z);
            color = over(color,thisSeg_color);
            if (color.w >= .98f)
              break;

            last_t_end = thisSeg_t1*(1.f+1e-6f);
#endif
          }
        } else {
          // no space skipping - just integrate the whole
          // ray-worldbounds overlap segment
          float z;
          color = integrateRay(ray,ils_t0,z);
        }
        color = over(color,bgColor);
      }
      accumColor += color;
    }

    uint64_t clock_end = clock();
    if (lp.render.heatMapEnabled > 0.f) {
      float t = (clock_end-clock_begin)*(lp.render.heatMapScale/spp);
      accumColor = over(vec4f(heatMap(t),.5f),accumColor);
    }

    int pixelID = threadIdx.x + owl::getLaunchDims().x*threadIdx.y;
    if (lp.accumID > 0)
      accumColor += vec4f(lp.accumBuffer[pixelID]);
    lp.accumBuffer[pixelID] = accumColor;
    accumColor *= (1.f/(lp.accumID+1));
    
    bool crossHairs = (owl::getLaunchIndex().x == owl::getLaunchDims().x/2
                       ||
                       owl::getLaunchIndex().y == owl::getLaunchDims().y/2
                       );
    if (crossHairs) accumColor = vec4f(1.f) - accumColor;
    
    lp.fbPointer[pixelID] = make_rgba(vec3f(accumColor*(1.f/spp)));
#if DUMP_FRAMES
    lp.fbDepth[pixelID]   = z;
#endif
  }
  
}
